#include "hip/hip_runtime.h"
/*
 * Modifications Copyright 2019 Graz University of Technology
 *
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/**********************************
 When updating a kernel or adding a new one,
 please compile the ptx file and commit it:
 nvcc -w -ptx -arch=sm_30 --std c++11 SystemDS.cu
 ***********************************/

#include <cfloat>
#include <cmath>
using uint = unsigned int;
#include <hip/hip_runtime.h>
#include <>

#include "utils.cuh"
#include "agg_ops.cuh"
#include "cum_scan.cuh"
#include "cum_sum.cuh"
#include "cum_prod.cuh"
#include "cum_min.cuh"
#include "cum_max.cuh"
#include "cum_sum_prod.cuh"

/**
 * This method performs an im2col operation on sparse input image
 *
 * @params inVal input val pointer
 * @params inRowPtr input row pointer
 * @params colInd input col index pointer
 * @param ret output matrix allocated on the GPU
 * @param NCHW  value of N*C*H*W
 * @param CHW value of C*H*W
 * @param HW value of H*W
 * @param W image height
 * @param R filter height
 * @param S filter width
 * @param P height of conv2d output
 * @param Q width of conv2d output
 * @param PQ value of P*Q
 * @param RS value of R*S
 * @param NPQ value of N*P*Q
 * @param stride_h stride height
 * @param stride_w stride width
 * @param pad_h padding height
 * @param pad_w padding width
 */
template<typename T>
__device__ void sparse_dense_im2col(T *inVal, int *inRowPtr, int *colInd,
		T *ret, int nnz, int N, int CHW, int HW, int W, int R, int S, int P,
		int Q, int PQ, int RS, int NPQ, int stride_h, int stride_w, int pad_h,
		int pad_w) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < nnz) {
		T value = inVal[tid];
		int n = 0;
		while (inRowPtr[n + 1] <= tid) {
			n++;
		}
		int chw = colInd[tid];
		int c = chw / HW;
		int hw = chw % HW;
		int h = hw / W;
		int w = hw % W;

		// Constraints: for(int r = 0; r < R; r++) { if(0 <= p && p < P && (h - r + pad_h) % stride_h == 0) { ... } }
		// Constraint 1: p >= 0 and p = (h - r + pad_h)  / stride_h
		// Therefore,  r <= h + pad_h
		// Constraint 2: p < P and p = (h - r + pad_h)  / stride_h
		// Therefore,  h + pad_h - P*stride_h < r
		// Math.max(0, h + pad_h - P*stride_h + 1) <= r <= Math.min(R-1, h + pad_h)
		int rMin = max(0, h + pad_h - P * stride_h + 1);
		int rMax = min(R - 1, h + pad_h);
		int sMin = max(0, w + pad_w - Q * stride_w + 1);
		int sMax = min(S - 1, w + pad_w);
		// Constraint 3: (h - r + pad_h) % stride_h == 0
		while ((h - rMin + pad_h) % stride_h != 0 && rMin <= rMax)
			rMin++;
		while ((w - sMin + pad_w) % stride_w != 0 && sMin <= sMax)
			sMin++;

		for (int r = rMin; r <= rMax; r += stride_h) {
			// Only append value if h == h, where h = (r - pad_h) + p*stride_h and 0 <= p < P
			// Therefore, p = (h - r + pad_h)  / stride_h. Use the same logic for q.
			int p = (h - r + pad_h) / stride_h;
			int npQ = n * PQ + p * Q;
			int outRowIndex = c * RS + r * S;
			for (int s = sMin; s <= sMax; s += stride_w) {
				int q = (w - s + pad_w) / stride_w;
				// chw -> [crs, npq]
				ret[(outRowIndex + s) * NPQ + npQ + q] = value;
			}
		}
	}
}

extern "C" __global__ void sparse_dense_im2col_d(double *inVal, int *inRowPtr,
		int *colInd, double *ret, int nnz, int N, int CHW, int HW, int W, int R,
		int S, int P, int Q, int PQ, int RS, int NPQ, int stride_h,
		int stride_w, int pad_h, int pad_w) {
	sparse_dense_im2col(inVal, inRowPtr, colInd, ret, nnz, N, CHW, HW, W, R, S,
			P, Q, PQ, RS, NPQ, stride_h, stride_w, pad_h, pad_w);
}

extern "C" __global__ void sparse_dense_im2col_f(float *inVal, int *inRowPtr,
		int *colInd, float *ret, int nnz, int N, int CHW, int HW, int W, int R,
		int S, int P, int Q, int PQ, int RS, int NPQ, int stride_h,
		int stride_w, int pad_h, int pad_w) {
	sparse_dense_im2col(inVal, inRowPtr, colInd, ret, nnz, N, CHW, HW, W, R, S,
			P, Q, PQ, RS, NPQ, stride_h, stride_w, pad_h, pad_w);
}

/**
 * This method performs an im2col operation on dense input image
 *
 * @param input input matrix allocated on the GPU
 * @param ret output matrix allocated on the GPU
 * @param NCHW  value of N*C*H*W
 * @param CHW value of C*H*W
 * @param HW value of H*W
 * @param W image height
 * @param R filter height
 * @param S filter width
 * @param P height of conv2d output
 * @param Q width of conv2d output
 * @param PQ value of P*Q
 * @param RS value of R*S
 * @param NPQ value of N*P*Q
 * @param stride_h stride height
 * @param stride_w stride width
 * @param pad_h padding height
 * @param pad_w padding width
 */
template<typename T>
__device__ void dense_dense_im2col(T *input, T *ret, int NCHW, int CHW, int HW,
		int W, int R, int S, int P, int Q, int PQ, int RS, int NPQ,
		int stride_h, int stride_w, int pad_h, int pad_w) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < NCHW) {
		T value = input[tid];
		int n = tid / CHW;
		int chw = tid % CHW;
		int c = chw / HW;
		int hw = chw % HW;
		int h = hw / W;
		int w = hw % W;

		// Constraints: for(int r = 0; r < R; r++) { if(0 <= p && p < P && (h - r + pad_h) % stride_h == 0) { ... } }
		// Constraint 1: p >= 0 and p = (h - r + pad_h)  / stride_h
		// Therefore,  r <= h + pad_h
		// Constraint 2: p < P and p = (h - r + pad_h)  / stride_h
		// Therefore,  h + pad_h - P*stride_h < r
		// Math.max(0, h + pad_h - P*stride_h + 1) <= r <= Math.min(R-1, h + pad_h)
		int rMin = max(0, h + pad_h - P * stride_h + 1);
		int rMax = min(R - 1, h + pad_h);
		int sMin = max(0, w + pad_w - Q * stride_w + 1);
		int sMax = min(S - 1, w + pad_w);
		// Constraint 3: (h - r + pad_h) % stride_h == 0
		while ((h - rMin + pad_h) % stride_h != 0 && rMin <= rMax)
			rMin++;
		while ((w - sMin + pad_w) % stride_w != 0 && sMin <= sMax)
			sMin++;

		for (int r = rMin; r <= rMax; r += stride_h) {
			// Only append value if h == h, where h = (r - pad_h) + p*stride_h and 0 <= p < P
			// Therefore, p = (h - r + pad_h)  / stride_h. Use the same logic for q.
			int p = (h - r + pad_h) / stride_h;
			int npQ = n * PQ + p * Q;
			int outRowIndex = c * RS + r * S;
			for (int s = sMin; s <= sMax; s += stride_w) {
				int q = (w - s + pad_w) / stride_w;
				// chw -> [crs, npq]
				ret[(outRowIndex + s) * NPQ + npQ + q] = value;
			}
		}
	}
}

extern "C" __global__ void dense_dense_im2col_d(double *input, double *ret,
		int NCHW, int CHW, int HW, int W, int R, int S, int P, int Q, int PQ,
		int RS, int NPQ, int stride_h, int stride_w, int pad_h, int pad_w) {
	dense_dense_im2col(input, ret, NCHW, CHW, HW, W, R, S, P, Q, PQ, RS, NPQ,
			stride_h, stride_w, pad_h, pad_w);
}

extern "C" __global__ void dense_dense_im2col_f(float *input, float *ret,
		int NCHW, int CHW, int HW, int W, int R, int S, int P, int Q, int PQ,
		int RS, int NPQ, int stride_h, int stride_w, int pad_h, int pad_w) {
	dense_dense_im2col(input, ret, NCHW, CHW, HW, W, R, S, P, Q, PQ, RS, NPQ,
			stride_h, stride_w, pad_h, pad_w);
}

/**
 * This method performs a reorg operation of matrix with dimensions [K, NPQ]
 * and returns a matrix with dimensions [N, KPQ]
 *
 * @param knpqPtr input matrix allocated on the GPU
 * @param ret output matrix allocated on the GPU
 * @param NKPQ length of input and output matrix
 * @param NPQ the number of columns of input matrix
 * @param KPQ the number of columns of output matrix
 * @param PQ value of P*Q
 */
template<typename T>
__device__ void reorg_knpq(T *knpqPtr, T *ret, int NKPQ, int NPQ, int KPQ,
		int PQ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < NKPQ) {
		int k = tid / NPQ;
		int npq = tid % NPQ;
		int n = npq / PQ;
		int pq = npq % PQ;
		ret[n * KPQ + k * PQ + pq] = knpqPtr[tid];
	}
}

extern "C" __global__ void reorg_knpq_d(double *knpqPtr, double *ret, int NKPQ,
		int NPQ, int KPQ, int PQ) {
	reorg_knpq(knpqPtr, ret, NKPQ, NPQ, KPQ, PQ);
}

extern "C" __global__ void reorg_knpq_f(float *knpqPtr, float *ret, int NKPQ,
		int NPQ, int KPQ, int PQ) {
	reorg_knpq(knpqPtr, ret, NKPQ, NPQ, KPQ, PQ);
}

/**
 * Performs a slice operation where the input matrix is sparse and the output
 * matrix is dense.
 * This function avoids unnecessary sparse to dense conversion of the input
 * matrix.
 * Parallelization: rows of output matrix.
 *
 * @params inVal input val pointer
 * @params inRowPtr input row pointer
 * @params colInd input col index pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param retClen number of columns of output matrix
 */
template<typename T>
__device__ void slice_sparse_dense_row(T *inVal, int *inRowPtr, int *colInd,
		T *ret, int rl, int ru, int cl, int cu, int retClen) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int rowIndex = index + rl;
	if (rowIndex <= ru) {
		/*
		 * TODO: Alternative approach: use dynamic parallelism. We are skipping this
		 *for now to avoid
		 * the complexity of two-step separate compilation and linking process.
		 *
		 * extern "C"
		 * __global__ void slice_sparse_dense_row_helper(double* inVal, int*
		 *inRowPtr, int* colInd, double* ret,
		 *     int rl, int ru, int cl, int cu, int retClen, int start, int end, int
		 *index) {
		 *  int i = blockIdx.x * blockDim.x + threadIdx.x + start;
		 * 	// Only slice if the index falls into the given range
		 * 	if(i < end && cl <= colInd[i] && colInd[i] <= cu) {
		 * 		ret[ index*retClen + (colInd[i] - cl) ] = inVal[i];
		 * 	}
		 * }
		 *
		 * int size = inRowPtr[rowIndex+1] - inRowPtr[rowIndex];
		 * double numThreads = (double)min(size, MAX_NUM_THREADS_CHILD_KERNEL);
		 * slice_sparse_dense_row_helper
		 * <<< ceil(numThreads/MAX_NUM_THREADS_CHILD_KERNEL), MAX_NUM_THREADS_CHILD_KERNEL>>>
		 * (inVal, inRowPtr, colInd, ret, rl, ru, cl, cu, retClen, inRowPtr[rowIndex],
		 *	inRowPtr[rowIndex+1], index);
		 *
		 * Two-step compilation and linking process in JCudaKernels's constructor:
		 * hiprtcLinkAddFile(linkState, hipJitInputType.hipJitInputLibrary,
		 * "/usr/local/cuda/lib64/libcudadevrt.a", jitOptions);
		 */
		// Iterate over elements of the row 'rowIndex'.
		for (int i = inRowPtr[rowIndex]; i < inRowPtr[rowIndex + 1]; i++) {
			// Only slice if the index falls into the given range
			if (cl <= colInd[i] && colInd[i] <= cu) {
				ret[index * retClen + (colInd[i] - cl)] = inVal[i];
			}
		}
	}
}

extern "C" __global__ void slice_sparse_dense_row_d(double *inVal,
		int *inRowPtr, int *colInd, double *ret, int rl, int ru, int cl, int cu,
		int retClen) {
	slice_sparse_dense_row(inVal, inRowPtr, colInd, ret, rl, ru, cl, cu,
			retClen);
}

extern "C" __global__ void slice_sparse_dense_row_f(float *inVal, int *inRowPtr,
		int *colInd, float *ret, int rl, int ru, int cl, int cu, int retClen) {
	slice_sparse_dense_row(inVal, inRowPtr, colInd, ret, rl, ru, cl, cu,
			retClen);
}

/**
 * Performs a slice operation where the input matrix is sparse and the output
 * matrix is dense.
 * This function avoids unnecessary sparse to dense conversion of the input
 * matrix.
 * Parallelization: subset of number of non-zeroes of input matrix.
 *
 * @params inVal input val pointer
 * @params inRowPtr input row pointer
 * @params colInd input col index pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param retClen number of columns of output matrix
 */
template<typename T>
__device__ void slice_sparse_dense_nnz(T *inVal, int *inRowPtr, int *colInd,
		T *ret, int rl, int ru, int cl, int cu, int retClen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = tid + inRowPtr[rl];

	// Only slice if the index falls into the given range
	if (i < inRowPtr[ru + 1] && cl <= colInd[i] && colInd[i] <= cu) {
		// Find the row index for corresponding non-zero value 'i'.
		int rowIndex = rl;
		while (inRowPtr[rowIndex + 1] <= i) {
			rowIndex++;
		}
		ret[(rowIndex - rl) * retClen + (colInd[i] - cl)] = inVal[i];
	}
}

extern "C" __global__ void slice_sparse_dense_nnz_d(double *inVal,
		int *inRowPtr, int *colInd, double *ret, int rl, int ru, int cl, int cu,
		int retClen) {
	slice_sparse_dense_nnz(inVal, inRowPtr, colInd, ret, rl, ru, cl, cu,
			retClen);
}

extern "C" __global__ void slice_sparse_dense_nnz_f(float *inVal, int *inRowPtr,
		int *colInd, float *ret, int rl, int ru, int cl, int cu, int retClen) {
	slice_sparse_dense_nnz(inVal, inRowPtr, colInd, ret, rl, ru, cl, cu,
			retClen);
}

/**
 * Performs a slice operation where the input matrix is dense and the output
 * matrix is dense.
 *
 * @params in dense input pointer
 * @params ret dense output pointer
 * @param rl row lower
 * @param ru row upper
 * @param cl column lower
 * @param cu column upper
 * @param inClen number of columns of input matrix
 * @param retRlen number of rows of output matrix
 * @param retClen number of columns of output matrix
 */
template<typename T>
__device__ void slice_dense_dense(T *in, T *ret, int rl, int ru, int cl, int cu,
		int inClen, int retRlen, int retClen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / retClen;
	int iy = tid % retClen;
	if (ix < retRlen && iy < retClen) {
		int inIndex = (ix + rl) * inClen + cl + iy;
		ret[tid] = in[inIndex];
	}
}

extern "C" __global__ void slice_dense_dense_d(double *in, double *ret, int rl,
		int ru, int cl, int cu, int inClen, int retRlen, int retClen) {
	slice_dense_dense(in, ret, rl, ru, cl, cu, inClen, retRlen, retClen);
}

extern "C" __global__ void slice_dense_dense_f(float *in, float *ret, int rl,
		int ru, int cl, int cu, int inClen, int retRlen, int retClen) {
	slice_dense_dense(in, ret, rl, ru, cl, cu, inClen, retRlen, retClen);
}

/**
 * Does a copy of upper to lower triangle of the given matrix
 * @param ret the input and output array allocated on the GPU
 * @param dim the number of rows of the square matrix ret
 * @param N total number of elements of the matrix
 */
template<typename T>
__device__ void copy_u2l_dense(T *ret, int dim, int N) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / dim;
	int iy = tid % dim;
	int id_dest = iy * dim + ix;
	if (iy > ix && id_dest < N) {
		// TODO: Potential to reduce the number of threads by half
		int id_src = tid;
		ret[id_dest] = ret[id_src];
	}
}

extern "C" __global__ void copy_u2l_dense_d(double *ret, int dim, int N) {
	copy_u2l_dense(ret, dim, N);
}

extern "C" __global__ void copy_u2l_dense_f(float *ret, int dim, int N) {
	copy_u2l_dense(ret, dim, N);
}

// Use this method in templates to fetch the maximum value for a given datatype
template<typename T>
__forceinline__ __device__ T MAX() {
	return T();
}
template<>
__forceinline__ __device__ float MAX<float>() {
	return FLT_MAX;
}
template<>
__forceinline__ __device__ double MAX<double>() {
	return DBL_MAX;
}

// op = {0=plus, 1=minus, 2=multiply, 3=divide, 4=power,
// 5=less, 6=lessequal, 7=greater, 8=greaterequal, 9=equal, 10=notequal,
// 11=min, 12=max, 13=and, 14=or, 15=minus1multiply, 16=minusnz,
// 17=modulus, 18=integer division}
template<typename T>
__forceinline__ __device__ T binaryOp(T x, T y, int op) {
	switch (op) {
	case 0:
		return x + y;
	case 1:
		return x - y;
	case 2:
		return x * y;
	case 3:
		return x / y;
	case 4:
		return pow(x, y);
	case 5:
		return (x < y) == 0 ? 0.0 : 1.0;
	case 6:
		return (x <= y) == 0 ? 0.0 : 1.0;
	case 7:
		return (x > y) == 0 ? 0.0 : 1.0;
	case 8:
		return (x >= y) == 0 ? 0.0 : 1.0;
	case 9:
		return (x == y) == 0 ? 0.0 : 1.0;
	case 10:
		return (x != y) == 0 ? 0.0 : 1.0;
	case 11:
		return min(x, y);
	case 12:
		return max(x, y);
	case 13:
		return ((int) llrint(x) & (int) llrint(y)) == 0 ? 0.0 : 1.0;
	case 14:
		return ((int) llrint(x) | (int) llrint(y)) == 0 ? 0.0 : 1.0;
	case 15:
		return 1 - x * y;
	case 16:
		return (x != 0.0 ? x - y : 0.0);
	case 17: {
		if (y == 0.0 || y == -0.0) {
			return nan("");
		}
		T v = x / y;
		// Check for v being NaN (v != v) or if it is infinity
		if (isnan(v) || isinf(v)) {
			return v;
		} else {
			v = floor(v);
		}
		return x - v * y;
	}
	case 18: {
		T v = x / y;
		if (isnan(v) || isinf(v)) {
			return v;
		} else {
			return floor(v);
		}
	}
	default:
		return MAX<T>();
	}
}

/**
 * Performs forward pass for relu: ret = max(A, 0)
 *
 * @param A input array allocated on the GPU
 * @param ret output array allocated on the GPU
 * @param rlen the number of rows
 * @param clen the number of columns
 */
template<typename T>
__device__ void relu(T *A, T *ret, int rlen, int clen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if (ix < rlen && iy < clen) {
		ret[tid] = max(0.0, A[tid]);
	}
}

extern "C" __global__ void relu_d(double *A, double *ret, int rlen, int clen) {
	relu(A, ret, rlen, clen);
}

extern "C" __global__ void relu_f(float *A, float *ret, int rlen, int clen) {
	relu(A, ret, rlen, clen);
}

/**
 * This method computes the backpropagation errors for previous layer of relu
 * operation
 *
 * @param X input activation array allocated on the GPU
 * @param dout errors from previous layer
 * @param ret output array allocated on the GPU
 * @param rlen the number of rows
 * @param clen the number of columns
 */
template<typename T>
__device__ void relu_backward(T *X, T *dout, T *ret, int rlen, int clen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if (ix < rlen && iy < clen) {
		ret[tid] = X[tid] > 0 ? dout[tid] : 0;
	}
}

extern "C" __global__ void relu_backward_d(double *X, double *dout, double *ret,
		int rlen, int clen) {
	relu_backward(X, dout, ret, rlen, clen);
}

extern "C" __global__ void relu_backward_f(float *X, float *dout, float *ret,
		int rlen, int clen) {
	relu_backward(X, dout, ret, rlen, clen);
}

/**
 * Performs inplace addition: ret += input
 *
 * @param input rhs input array allocated on the GPU
 * @param ret the input and output array allocated on the GPU
 * @param rlen the number of rows
 * @param clen the number of columns
 */
template<typename T>
__device__ void inplace_add(T *input, T *ret, int rlen, int clen) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if (ix < rlen && iy < clen) {
		ret[tid] += input[tid];
	}
}

extern "C" __global__ void inplace_add_d(double *input, double *ret, int rlen,
		int clen) {
	inplace_add(input, ret, rlen, clen);
}

extern "C" __global__ void inplace_add_f(float *input, float *ret, int rlen,
		int clen) {
	inplace_add(input, ret, rlen, clen);
}

// Performs the operation corresponding to the DML script:
// ones = matrix(1, rows=1, cols=Hout*Wout)
// output = input + matrix(bias %*% ones, rows=1, cols=F*Hout*Wout)
// This operation is often followed by conv2d and hence we have introduced
// bias_add(input, bias) built-in function
template<typename T>
__device__ void bias_add(T *input, T *bias, T *ret, int rlen, int clen,
		int PQ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if (ix < rlen && iy < clen) {
		int biasIndex = iy / PQ;
		ret[tid] = input[tid] + bias[biasIndex];
	}
}

extern "C" __global__ void bias_add_d(double *input, double *bias, double *ret,
		int rlen, int clen, int PQ) {
	bias_add(input, bias, ret, rlen, clen, PQ);
}

extern "C" __global__ void bias_add_f(float *input, float *bias, float *ret,
		int rlen, int clen, int PQ) {
	bias_add(input, bias, ret, rlen, clen, PQ);
}

// Performs the operation "ret <- A + alpha*B", where B is a vector
template<typename T>
__device__ void daxpy_matrix_vector(T *A, T *B, double alpha, T *ret, int rlenA,
		int clenA, int rlenB, int clenB) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clenA;
	int iy = tid % clenA;
	if (ix < rlenA && iy < clenA) {
		int index = ix * clenA + iy;
		if (rlenB == 1) {
			ret[index] = A[index] + alpha * B[iy];
		} else {
			ret[index] = A[index] + alpha * B[ix];
		}
	}
}

extern "C" __global__ void daxpy_matrix_vector_d(double *A, double *B,
		double alpha, double *ret, int rlenA, int clenA, int rlenB, int clenB) {
	daxpy_matrix_vector(A, B, alpha, ret, rlenA, clenA, rlenB, clenB);
}

extern "C" __global__ void daxpy_matrix_vector_f(float *A, float *B,
		double alpha, float *ret, int rlenA, int clenA, int rlenB, int clenB) {
	daxpy_matrix_vector(A, B, alpha, ret, rlenA, clenA, rlenB, clenB);
}

// Performs similar operation as bias_add except elementwise multiplication
// instead of add
template<typename T>
__device__ void bias_multiply(T *input, T *bias, T *ret, int rlen, int clen,
		int PQ) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / clen;
	int iy = tid % clen;
	if (ix < rlen && iy < clen) {
		int biasIndex = iy / PQ;
		ret[tid] = input[tid] * bias[biasIndex];
	}
}

extern "C" __global__ void bias_multiply_d(double *input, double *bias,
		double *ret, int rlen, int clen, int PQ) {
	bias_multiply(input, bias, ret, rlen, clen, PQ);
}

extern "C" __global__ void bias_multiply_f(float *input, float *bias,
		float *ret, int rlen, int clen, int PQ) {
	bias_multiply(input, bias, ret, rlen, clen, PQ);
}

/**
 * Performs a binary cellwise arithmetic operation on 2 matrices.
 * Either both matrices are of equal size or one of them is a vector or both
 * are.
 * @param A                 first input matrix allocated on GPU
 * @param B                 second input matrix allocated on GPU
 * @param C                 output allocated on GPU
 * @param maxRlen           maximum of the row lengths of A and B
 * @param maxClen           maximum of the column lengths of A and B
 * @param vectorAStatus     if A is a row vector, column vector or neither
 * @param vectorBStatus     if B is a row vector, column vector or neither
 * @param op                the numeric code of the arithmetic operation to
 * perform
 *
 */
template<typename T>
__device__ void matrix_matrix_cellwise_op(T *A, T *B, T *C, int maxRlen,
		int maxClen, int vectorAStatus, int vectorBStatus, int op) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / maxClen;
	int iy = tid % maxClen;

	if (ix < maxRlen && iy < maxClen) {
		int outIndex = ix * maxClen + iy;
		int aIndex = outIndex;
		int bIndex = outIndex;
		if (vectorAStatus == 1)
			aIndex = ix;  // clen == 1
		else if (vectorAStatus == 2)
			aIndex = iy;  // rlen == 1
		if (vectorBStatus == 1)
			bIndex = ix;  // clen == 1
		else if (vectorBStatus == 2)
			bIndex = iy;  // rlen == 1
		C[outIndex] = binaryOp(A[aIndex], B[bIndex], op);
		// printf("C[%d] = A[%d](%f) B[%d](%f) (%d %d)\n", outIndex, aIndex,
		// A[aIndex], bIndex,  B[bIndex], (ix+1), (iy+1));
		__syncthreads();
	}
}

extern "C" __global__ void matrix_matrix_cellwise_op_d(double *A, double *B,
		double *C, int maxRlen, int maxClen, int vectorAStatus,
		int vectorBStatus, int op) {
	matrix_matrix_cellwise_op(A, B, C, maxRlen, maxClen, vectorAStatus,
			vectorBStatus, op);
}

extern "C" __global__ void matrix_matrix_cellwise_op_f(float *A, float *B,
		float *C, int maxRlen, int maxClen, int vectorAStatus,
		int vectorBStatus, int op) {
	matrix_matrix_cellwise_op(A, B, C, maxRlen, maxClen, vectorAStatus,
			vectorBStatus, op);
}

/**
 * Performs an arithmetic operation between a matrix and a scalar.
 * C = s op A or C = A op s (where A is the matrix, s is the scalar and op is
 * the operation)
 * @param A             input matrix allocated on GPU
 * @param scalar        scalar input
 * @param C             output matrix allocated on GPU
 * @param size          number of elements in matrix A
 * @param op            number code of the arithmetic operation to perform
 * @param isLeftScalar  whether the scalar is on the left side
 */
template<typename T>
__device__ void matrix_scalar_op(T *A, T scalar, T *C, int size, int op,
		int isLeftScalar) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		if (isLeftScalar) {
			C[index] = binaryOp(scalar, A[index], op);
		} else {
			C[index] = binaryOp(A[index], scalar, op);
		}
	}
	__syncthreads();
}

extern "C" __global__ void matrix_scalar_op_d(double *A, double scalar,
		double *C, int size, int op, int isLeftScalar) {
	matrix_scalar_op(A, scalar, C, size, op, isLeftScalar);
}

extern "C" __global__ void matrix_scalar_op_f(float *A, double scalar, float *C,
		int size, int op, int isLeftScalar) {
	matrix_scalar_op(A, (float) scalar, C, size, op, isLeftScalar);
}

/**
 * Sets all elements (fills) of a double array of given length with a given
 * scalar value
 * @param A         array to be filled
 * @param scalar    value to fill array with
 * @param lenA      length of array A
 */
template<typename T>
__device__ void fill(T *A, T scalar, int lenA) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < lenA) {
		A[index] = scalar;
	}
}

extern "C" __global__ void fill_d(double *A, double scalar, int lenA) {
	fill(A, scalar, lenA);
}

extern "C" __global__ void fill_f(float *A, double scalar, int lenA) {
	fill(A, (float) scalar, lenA);
}

/**
 * Appends Matrix B to the right side of Matrix A into a new matrix C
 *         | 1 2 3 4 |   | 8 8 8 |     | 1 2 3 4 8 8 8 |
 * cbind ( | 9 8 7 6 | , | 7 7 7 | ) = | 9 8 7 6 7 7 7 |
 *         | 4 3 2 1 |   | 9 9 9 |     | 4 3 2 1 9 9 9 |
 * @param A      input matrix A allocated on the GPU
 * @param B      input matrix B allocated on the GPU
 * @param C      input matrix C allocated on the GPU
 * @param rowsA  rows in A
 * @param colsA  columns in A
 * @param rowsB  rows in B
 * @param colsB  columns in B
 */
template<typename T>
__device__ void cbind(T *A, T *B, T *C, int rowsA, int colsA, int rowsB,
		int colsB) {
	int maxClen = max(colsA, colsB);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / maxClen;
	int iy = tid % maxClen;

	int colsC = colsA + colsB;

	// Copy an element of A into C into the appropriate location
	if (ix < rowsA && iy < colsA) {
		T elemA = A[ix * colsA + iy];
		C[ix * colsC + iy] = elemA;
	}

	// Copy an element of B into C into the appropriate location
	if (ix < rowsB && iy < colsB) {
		T elemB = B[ix * colsB + iy];
		C[ix * colsC + (iy + colsA)] = elemB;
	}
}

extern "C" __global__ void cbind_d(double *A, double *B, double *C, int rowsA,
		int colsA, int rowsB, int colsB) {
	cbind(A, B, C, rowsA, colsA, rowsB, colsB);
}

extern "C" __global__ void cbind_f(float *A, float *B, float *C, int rowsA,
		int colsA, int rowsB, int colsB) {
	cbind(A, B, C, rowsA, colsA, rowsB, colsB);
}

/**
 * Appends Matrix B to the bottom of Matrix A into a new matrix C
 *         | 2 3 4 |   | 8 8 8 |     | 2 3 4 |
 * rbind ( | 8 7 6 | , | 7 7 7 | ) = | 8 7 6 |
 *         | 3 2 1 |                 | 3 2 1 |
 | 8 8 8 |
 | 7 7 7 |
 * @param A      input matrix A allocated on the GPU
 * @param B      input matrix B allocated on the GPU
 * @param C      input matrix C allocated on the GPU
 * @param rowsA  rows in A
 * @param colsA  columns in A
 * @param rowsB  rows in B
 * @param colsB  columns in B
 */
template<typename T>
__device__ void rbind(T *A, T *B, T *C, int rowsA, int colsA, int rowsB,
		int colsB) {
	int maxClen = max(colsA, colsB);
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int ix = tid / maxClen;
	int iy = tid % maxClen;

	int colsC = colsA;

	// Copy an element of A into C into the appropriate location
	if (ix < rowsA && iy < colsA) {
		T elemA = A[ix * colsA + iy];
		C[ix * colsC + iy] = elemA;
	}

	// Copy an element of B into C into the appropriate location
	if (ix < rowsB && iy < colsB) {
		T elemB = B[ix * colsB + iy];
		C[(ix + rowsA) * colsC + iy] = elemB;
	}
}

extern "C" __global__ void rbind_d(double *A, double *B, double *C, int rowsA,
		int colsA, int rowsB, int colsB) {
	rbind(A, B, C, rowsA, colsA, rowsB, colsB);
}

extern "C" __global__ void rbind_f(float *A, float *B, float *C, int rowsA,
		int colsA, int rowsB, int colsB) {
	rbind(A, B, C, rowsA, colsA, rowsB, colsB);
}


/**
 * Does a reduce operation over all elements of the array.
 * This method has been adapted from the Reduction sample in the NVIDIA CUDA
 * Samples (v8.0)
 * and the Reduction example available through jcuda.org
 * When invoked initially, all blocks partly compute the reduction operation
 * over the entire array
 * and writes it to the output/temporary array. A second invokation needs to
 * happen to get the
 * reduced value.
 * The number of threads, blocks and amount of shared memory is calculated in a
 * specific way.
 * Please refer to the NVIDIA CUDA Sample or the SystemDS code that invokes this
 * method to see
 * how its done.
 * The template-ized version of this function is similar to what is found in
 * NVIDIA CUB
 *
 * @param ReductionOp       Type of the functor object that implements the
 * reduction operation
 */
template<typename ReductionOp, typename T>
__device__ void reduce(T *g_idata, ///< input data stored in device memory (of size n)
		T *g_odata, ///< output/temporary array stored in device memory (of size n)
		unsigned int n,  ///< size of the input and temporary/output arrays
		ReductionOp reduction_op, ///< Reduction operation to perform (functor object)
		T initialValue)    ///< initial value for the reduction variable
{
	auto sdata = shared_memory_proxy<T>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
	unsigned int gridSize = blockDim.x * 2 * gridDim.x;

	T v = initialValue;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n) {
		v = reduction_op(v, g_idata[i]);
		// ensure we don't read out of bounds
		if (i + blockDim.x < n)
			v = reduction_op(v, g_idata[i + blockDim.x]);
		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = v;
	__syncthreads();

	// do reduction in shared mem
	if (blockDim.x >= 1024) {
		if (tid < 512) {
			sdata[tid] = v = reduction_op(v, sdata[tid + 512]);
		}
		__syncthreads();
	}
	if (blockDim.x >= 512) {
		if (tid < 256) {
			sdata[tid] = v = reduction_op(v, sdata[tid + 256]);
		}
		__syncthreads();
	}
	if (blockDim.x >= 256) {
		if (tid < 128) {
			sdata[tid] = v = reduction_op(v, sdata[tid + 128]);
		}
		__syncthreads();
	}
	if (blockDim.x >= 128) {
		if (tid < 64) {
			sdata[tid] = v = reduction_op(v, sdata[tid + 64]);
		}
		__syncthreads();
	}

	if (tid < 32) {
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		volatile T *smem = sdata;
		if (blockDim.x >= 64) {
			smem[tid] = v = reduction_op(v, smem[tid + 32]);
		}
		if (blockDim.x >= 32) {
			smem[tid] = v = reduction_op(v, smem[tid + 16]);
		}
		if (blockDim.x >= 16) {
			smem[tid] = v = reduction_op(v, smem[tid + 8]);
		}
		if (blockDim.x >= 8) {
			smem[tid] = v = reduction_op(v, smem[tid + 4]);
		}
		if (blockDim.x >= 4) {
			smem[tid] = v = reduction_op(v, smem[tid + 2]);
		}
		if (blockDim.x >= 2) {
			smem[tid] = v = reduction_op(v, smem[tid + 1]);
		}
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

/**
 * Does a reduce (sum) over each row of the array.
 * This kernel must be launched with as many blocks as there are rows.
 * The intuition for this kernel is that each block does a reduction over a
 * single row.
 * The maximum number of blocks that can launched (as of compute capability 3.0)
 * is 2^31 - 1
 * This works out fine for SystemDS, since the maximum elements in a Java array
 * can be 2^31 - c (some small constant)
 * If the matrix is "fat" and "short", i.e. there are small number of rows and a
 * large number of columns,
 * there could be under-utilization of the hardware.
 * The template-ized version of this function is similar to what is found in
 * NVIDIA CUB
 * @param ReductionOp       Type of the functor object that implements the
 * reduction operation
 * @param AssignmentOp      Type of the functor object that is used to modify
 * the value before writing it to its final location in global memory for each
 * row
 */
template<typename ReductionOp, typename AssignmentOp, typename T>
__device__ void reduce_row(T *g_idata, ///< input data stored in device memory (of size rows*cols)
		T *g_odata,  ///< output/temporary array store in device memory (of size
		/// rows*cols)
		unsigned int rows,  ///< rows in input and temporary/output arrays
		unsigned int cols,  ///< columns in input and temporary/output arrays
		ReductionOp reduction_op, ///< Reduction operation to perform (functor object)
		AssignmentOp assignment_op, ///< Operation to perform before assigning this
		/// to its final location in global memory for
		/// each row
		T initialValue)  ///< initial value for the reduction variable
{
	auto sdata = shared_memory_proxy<T>();

	// one block per row
	if (blockIdx.x >= rows) {
		return;
	}

	unsigned int block = blockIdx.x;
	unsigned int tid = threadIdx.x;
	unsigned int i = tid;
	unsigned int block_offset = block * cols;

	T v = initialValue;
	while (i < cols) {
		v = reduction_op(v, g_idata[block_offset + i]);
		i += blockDim.x;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = v;
	__syncthreads();

	// do reduction in shared mem
	if (blockDim.x >= 1024) {
		if (tid < 512) {
			sdata[tid] = v = reduction_op(v, sdata[tid + 512]);
		}
		__syncthreads();
	}
	if (blockDim.x >= 512) {
		if (tid < 256) {
			sdata[tid] = v = reduction_op(v, sdata[tid + 256]);
		}
		__syncthreads();
	}
	if (blockDim.x >= 256) {
		if (tid < 128) {
			sdata[tid] = v = reduction_op(v, sdata[tid + 128]);
		}
		__syncthreads();
	}
	if (blockDim.x >= 128) {
		if (tid < 64) {
			sdata[tid] = v = reduction_op(v, sdata[tid + 64]);
		}
		__syncthreads();
	}

	if (tid < 32) {
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		volatile T *smem = sdata;
		if (blockDim.x >= 64) {
			smem[tid] = v = reduction_op(v, smem[tid + 32]);
		}
		if (blockDim.x >= 32) {
			smem[tid] = v = reduction_op(v, smem[tid + 16]);
		}
		if (blockDim.x >= 16) {
			smem[tid] = v = reduction_op(v, smem[tid + 8]);
		}
		if (blockDim.x >= 8) {
			smem[tid] = v = reduction_op(v, smem[tid + 4]);
		}
		if (blockDim.x >= 4) {
			smem[tid] = v = reduction_op(v, smem[tid + 2]);
		}
		if (blockDim.x >= 2) {
			smem[tid] = v = reduction_op(v, smem[tid + 1]);
		}
	}

	// write result for this block to global mem, modify it with assignment op
	if (tid == 0)
		g_odata[block] = assignment_op(sdata[0]);
}

/**
 * Does a column wise reduction.
 * The intuition is that there are as many global threads as there are columns
 * Each global thread is responsible for a single element in the output vector
 * This of course leads to a under-utilization of the GPU resources.
 * For cases, where the number of columns is small, there can be unused SMs
 *
 * The template-ized version of this function is similar to what is found in
 * NVIDIA CUB
 * @param ReductionOp       Type of the functor object that implements the
 * reduction operation
 * @param AssignmentOp      Type of the functor object that is used to modify
 * the value before writing it to its final location in global memory for each
 * column
 */
template<typename ReductionOp, typename AssignmentOp, typename T>
__device__ void reduce_col(T *g_idata, ///< input data stored in device memory (of size rows*cols)
		T *g_odata,  ///< output/temporary array store in device memory (of size rows*cols)
		unsigned int rows,  ///< rows in input and temporary/output arrays
		unsigned int cols,  ///< columns in input and temporary/output arrays
		ReductionOp reduction_op, ///< Reduction operation to perform (functor object)
		AssignmentOp assignment_op, ///< Operation to perform before assigning this
		/// to its final location in global memory for each column
		T initialValue)  ///< initial value for the reduction variable
{
	unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (global_tid >= cols) {
		return;
	}

	unsigned int i = global_tid;
	unsigned int grid_size = cols;
	T val = initialValue;

	while (i < rows * cols) {
		val = reduction_op(val, g_idata[i]);
		i += grid_size;
	}
	g_odata[global_tid] = assignment_op(val);
}


/**
 * Do a summation over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stored in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
template<typename T>
__device__ void reduce_sum(T *g_idata, T *g_odata, unsigned int n) {
	SumOp<T> op;
	reduce<SumOp<T>, T>(g_idata, g_odata, n, op, (T) 0.0);
}

extern "C" __global__ void reduce_sum_d(double *g_idata, double *g_odata,
		unsigned int n) {
	reduce_sum(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_sum_f(float *g_idata, float *g_odata,
		unsigned int n) {
	reduce_sum(g_idata, g_odata, n);
}


/**
 * Do a summation over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template<typename T>
__device__ void reduce_row_sum(T *g_idata, T *g_odata, unsigned int rows,
		unsigned int cols) {
	SumOp<T> op;
	IdentityOp<T> aop;
	reduce_row<SumOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op,
			aop, 0.0);
}

extern "C" __global__ void reduce_row_sum_d(double *g_idata, double *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_row_sum(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_row_sum_f(float *g_idata, float *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_row_sum(g_idata, g_odata, rows, cols);
}

/**
 * Do a summation over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template<typename T>
__device__ void reduce_col_sum(T *g_idata, T *g_odata, unsigned int rows,
		unsigned int cols) {
	SumOp<T> op;
	IdentityOp<T> aop;
	reduce_col<SumOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op,
			aop, (T) 0.0);
}

extern "C" __global__ void reduce_col_sum_d(double *g_idata, double *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_col_sum(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_col_sum_f(float *g_idata, float *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_col_sum(g_idata, g_odata, rows, cols);
}

/**
 * Do a max over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
template<typename T>
__device__ void reduce_max(T *g_idata, T *g_odata, unsigned int n) {
	MaxOp<T> op;
	reduce<MaxOp<T>, T>(g_idata, g_odata, n, op, -MAX<T>());
}

extern "C" __global__ void reduce_max_d(double *g_idata, double *g_odata,
		unsigned int n) {
	reduce_max(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_max_f(float *g_idata, float *g_odata,
		unsigned int n) {
	reduce_max(g_idata, g_odata, n);
}

/**
 * Do a max over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template<typename T>
__device__ void reduce_row_max(T *g_idata, T *g_odata, unsigned int rows,
		unsigned int cols) {
	MaxOp<T> op;
	IdentityOp<T> aop;
	reduce_row<MaxOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op,
			aop, -MAX<T>());
}

extern "C" __global__ void reduce_row_max_d(double *g_idata, double *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_row_max(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_row_max_f(float *g_idata, float *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_row_max(g_idata, g_odata, rows, cols);
}

/**
 * Do a max over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template<typename T>
__device__ void reduce_col_max(T *g_idata, T *g_odata, unsigned int rows,
		unsigned int cols) {
	MaxOp<T> op;
	IdentityOp<T> aop;
	reduce_col<MaxOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op,
			aop, -MAX<T>());
}

extern "C" __global__ void reduce_col_max_d(double *g_idata, double *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_col_max(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_col_max_f(float *g_idata, float *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_col_max(g_idata, g_odata, rows, cols);
}


/**
 * Do a min over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
template<typename T>
__device__ void reduce_min(T *g_idata, T *g_odata, unsigned int n) {
	MinOp<T> op;
	reduce<MinOp<T>, T>(g_idata, g_odata, n, op, MAX<T>());
}

extern "C" __global__ void reduce_min_d(double *g_idata, double *g_odata,
		unsigned int n) {
	reduce_min(g_idata, g_odata, n);
}

extern "C" __global__ void reduce_min_f(float *g_idata, float *g_odata,
		unsigned int n) {
	reduce_min(g_idata, g_odata, n);
}

/**
 * Do a min over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template<typename T>
__device__ void reduce_row_min(T *g_idata, T *g_odata, unsigned int rows,
		unsigned int cols) {
	MinOp<T> op;
	IdentityOp<T> aop;
	reduce_row<MinOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op,
			aop, MAX<T>());
}

extern "C" __global__ void reduce_row_min_d(double *g_idata, double *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_row_min(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_row_min_f(float *g_idata, float *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_row_min(g_idata, g_odata, rows, cols);
}

/**
 * Do a min over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template<typename T>
__device__ void reduce_col_min(T *g_idata, T *g_odata, unsigned int rows,
		unsigned int cols) {
	MinOp<T> op;
	IdentityOp<T> aop;
	reduce_col<MinOp<T>, IdentityOp<T>, T>(g_idata, g_odata, rows, cols, op,
			aop, MAX<T>());
}

extern "C" __global__ void reduce_col_min_d(double *g_idata, double *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_col_min(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_col_min_f(float *g_idata, float *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_col_min(g_idata, g_odata, rows, cols);
}




/**
 * Do a product over all elements of an array/matrix
 * @param g_idata   input data stored in device memory (of size n)
 * @param g_odata   output/temporary array stode in device memory (of size n)
 * @param n         size of the input and temporary/output arrays
 */
template<typename T>
__device__ void reduce_prod(T *g_idata, T *g_odata, unsigned int n) {
	ProductOp<T> op;
	reduce<ProductOp<T>, T>(g_idata, g_odata, n, op, (T) 1.0);
}

extern "C"
__global__ void reduce_prod_d(double *g_idata, double *g_odata,
		unsigned int n) {
	reduce_prod(g_idata, g_odata, n);
}

extern "C"
__global__ void reduce_prod_f(float *g_idata, float *g_odata,
		unsigned int n) {
	reduce_prod(g_idata, g_odata, n);
}

/**
 * Do a mean over all rows of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size rows)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template<typename T>
__device__ void reduce_row_mean(T *g_idata, T *g_odata, unsigned int rows,
		unsigned int cols) {
	SumOp<T> op;
	MeanOp<T> aop(cols);
	reduce_row<SumOp<T>, MeanOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
			(T) 0.0);
}

extern "C" __global__ void reduce_row_mean_d(double *g_idata, double *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_row_mean(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_row_mean_f(float *g_idata, float *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_row_mean(g_idata, g_odata, rows, cols);
}

/**
 * Do a mean over all columns of a matrix
 * @param g_idata   input matrix stored in device memory (of size rows * cols)
 * @param g_odata   output vector stored in device memory (of size cols)
 * @param rows      number of rows in input matrix
 * @param cols      number of columns in input matrix
 */
template<typename T>
__device__ void reduce_col_mean(T *g_idata, T *g_odata, unsigned int rows,
		unsigned int cols) {
	SumOp<T> op;
	MeanOp<T> aop(rows);
	reduce_col<SumOp<T>, MeanOp<T>, T>(g_idata, g_odata, rows, cols, op, aop,
			0.0);
}

extern "C" __global__ void reduce_col_mean_d(double *g_idata, double *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_col_mean(g_idata, g_odata, rows, cols);
}

extern "C" __global__ void reduce_col_mean_f(float *g_idata, float *g_odata,
		unsigned int rows, unsigned int cols) {
	reduce_col_mean(g_idata, g_odata, rows, cols);
}

/**
 * Do an exp over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_exp(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = exp(A[index]);
	}
}

extern "C" __global__ void matrix_exp_d(double *A, double *C,
		unsigned int size) {
	matrix_exp(A, C, size);
}

extern "C" __global__ void matrix_exp_f(float *A, float *C, unsigned int size) {
	matrix_exp(A, C, size);
}

/**
 * Do an sqrt over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_sqrt(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = sqrt(A[index]);
	}
}

extern "C" __global__ void matrix_sqrt_d(double *A, double *C,
		unsigned int size) {
	matrix_sqrt(A, C, size);
}

extern "C" __global__ void matrix_sqrt_f(float *A, float *C,
		unsigned int size) {
	matrix_sqrt(A, C, size);
}

/**
 * Do an round over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_round(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = (T) llround(A[index]);
	}
}

extern "C" __global__ void matrix_round_d(double *A, double *C,
		unsigned int size) {
	matrix_round(A, C, size);
}

extern "C" __global__ void matrix_round_f(float *A, float *C,
		unsigned int size) {
	matrix_round(A, C, size);
}

/**
 * Do an abs over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_abs(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = (T) fabs(A[index]);
	}
}

extern "C" __global__ void matrix_abs_d(double *A, double *C,
		unsigned int size) {
	matrix_abs(A, C, size);
}

extern "C" __global__ void matrix_abs_f(float *A, float *C, unsigned int size) {
	matrix_abs(A, C, size);
}

/**
 * Do an log over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_log(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = log(A[index]);
	}
}

extern "C" __global__ void matrix_log_d(double *A, double *C,
		unsigned int size) {
	matrix_log(A, C, size);
}

extern "C" __global__ void matrix_log_f(float *A, float *C, unsigned int size) {
	matrix_log(A, C, size);
}

/**
 * Do an floor over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_floor(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = floor(A[index]);
	}
}

extern "C" __global__ void matrix_floor_d(double *A, double *C,
		unsigned int size) {
	matrix_floor(A, C, size);
}

extern "C" __global__ void matrix_floor_f(float *A, float *C,
		unsigned int size) {
	matrix_floor(A, C, size);
}

/**
 * Do an ceil over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_ceil(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = ceil(A[index]);
	}
}

extern "C" __global__ void matrix_ceil_d(double *A, double *C,
		unsigned int size) {
	matrix_ceil(A, C, size);
}

extern "C" __global__ void matrix_ceil_f(float *A, float *C,
		unsigned int size) {
	matrix_ceil(A, C, size);
}

/**
 * Do an sin over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_sin(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = sin(A[index]);
	}
}

extern "C" __global__ void matrix_sin_d(double *A, double *C,
		unsigned int size) {
	matrix_sin(A, C, size);
}

extern "C" __global__ void matrix_sin_f(float *A, float *C, unsigned int size) {
	matrix_sin(A, C, size);
}

/**
 * Do an sinh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_sinh(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = sinh(A[index]);
	}
}

extern "C" __global__ void matrix_sinh_d(double *A, double *C,
		unsigned int size) {
	matrix_sinh(A, C, size);
}

extern "C" __global__ void matrix_sinh_f(float *A, float *C,
		unsigned int size) {
	matrix_sinh(A, C, size);
}

/**
 * Do an cos over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_cos(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = cos(A[index]);
	}
}

extern "C" __global__ void matrix_cos_d(double *A, double *C,
		unsigned int size) {
	matrix_cos(A, C, size);
}

extern "C" __global__ void matrix_cos_f(float *A, float *C, unsigned int size) {
	matrix_cos(A, C, size);
}

/**
 * Do an cosh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_cosh(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = cosh(A[index]);
	}
}

extern "C" __global__ void matrix_cosh_d(double *A, double *C,
		unsigned int size) {
	matrix_cosh(A, C, size);
}

extern "C" __global__ void matrix_cosh_f(float *A, float *C,
		unsigned int size) {
	matrix_cosh(A, C, size);
}

/**
 * Do an tan over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_tan(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = tan(A[index]);
	}
}

extern "C" __global__ void matrix_tan_d(double *A, double *C,
		unsigned int size) {
	matrix_tan(A, C, size);
}

extern "C" __global__ void matrix_tan_f(float *A, float *C, unsigned int size) {
	matrix_tan(A, C, size);
}

/**
 * Do an tanh over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_tanh(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = tanh(A[index]);
	}
}

extern "C" __global__ void matrix_tanh_d(double *A, double *C,
		unsigned int size) {
	matrix_tanh(A, C, size);
}

extern "C" __global__ void matrix_tanh_f(float *A, float *C,
		unsigned int size) {
	matrix_tanh(A, C, size);
}

/**
 * Do an asin over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_asin(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = asin(A[index]);
	}
}

extern "C" __global__ void matrix_asin_d(double *A, double *C,
		unsigned int size) {
	matrix_asin(A, C, size);
}

extern "C" __global__ void matrix_asin_f(float *A, float *C,
		unsigned int size) {
	matrix_asin(A, C, size);
}

/**
 * Do an acos over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_acos(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = acos(A[index]);
	}
}

extern "C" __global__ void matrix_acos_d(double *A, double *C,
		unsigned int size) {
	matrix_acos(A, C, size);
}

extern "C" __global__ void matrix_acos_f(float *A, float *C,
		unsigned int size) {
	matrix_acos(A, C, size);
}

/**
 * Do an atan over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_atan(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = atan(A[index]);
	}
}

extern "C" __global__ void matrix_atan_d(double *A, double *C,
		unsigned int size) {
	matrix_atan(A, C, size);
}

extern "C" __global__ void matrix_atan_f(float *A, float *C,
		unsigned int size) {
	matrix_atan(A, C, size);
}

/**
 * Do an sign over all the elements of a matrix
 * Assign -1, 0 or 1 depending on the element being negative, 0 or positive
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_sign(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		if (A[index] == 0.0) {
			C[index] = 0.0;
		} else {
			C[index] = copysign(1.0, A[index]);
		}
	}
}

extern "C" __global__ void matrix_sign_d(double *A, double *C,
		unsigned int size) {
	matrix_sign(A, C, size);
}

extern "C" __global__ void matrix_sign_f(float *A, float *C,
		unsigned int size) {
	matrix_sign(A, C, size);
}

/**
 * Do an sigmoid over all the elements of a matrix
 * @param A the input matrix (of length = size)
 * @param C the pre-allocated output matrix (of length = size)
 * @param siz the length of the input and output matrices
 */
template<typename T>
__device__ void matrix_sigmoid(T *A, T *C, unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		C[index] = 0.5 * tanh(0.5 * A[index]) + 0.5;
	}
}

extern "C" __global__ void matrix_sigmoid_d(double *A, double *C,
		unsigned int size) {
	matrix_sigmoid(A, C, size);
}

extern "C" __global__ void matrix_sigmoid_f(float *A, float *C,
		unsigned int size) {
	matrix_sigmoid(A, C, size);
}

template<typename T>
__device__ void prepare_lstm_input(T* sdsInput, T* cudnnInput, int N, int D,
		int TD, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		int n = index / TD;
		int td = index % TD;
		int t = td / D;
		int d = td % D;
		cudnnInput[t * N * D + n * D + d] = sdsInput[index];
	}
}

extern "C" __global__ void prepare_lstm_input_d(double* sdsInput,
		double* cudnnInput, int N, int D, int TD, int size) {
	prepare_lstm_input(sdsInput, cudnnInput, N, D, TD, size);
}

extern "C" __global__ void prepare_lstm_input_f(float* sdsInput,
		float* cudnnInput, int N, int D, int TD, int size) {
	prepare_lstm_input(sdsInput, cudnnInput, N, D, TD, size);
}

__device__ int swap_co(int offset) {
	return (offset < 2) ? offset : (offset == 2 ? 3 : 2);
}

__device__ void compute_lstm_weight_indexes(int index, int D, int M, int* ret) {
	// input: cbind(X_t, out_prev) => [N, D+M], weight: [D+M, 4M]
	// https://docs.nvidia.com/deeplearning/sdk/cudnn-developer-guide/index.html#hipdnnGetRNNLinLayerMatrixParams states that
	// Elements in each weight matrix are arranged in the row-major order, but the column-major format works !!
	// CuDNN gate order: i, f, c, o
	// CuDNN weight order: w_i, w_f, w_c, w_o, r_i, r_f, r_c, r_o
	// SystemDS weight order: i, f, o, c; TF weight order: i, c, f, o
	// SystemDS performs (X_t %*% W + out_prev %*% R) => [N, 4*M]
	int DM = D * M;
	int MM = M * M;
	int DM4 = DM * 4;
	int M4 = M * 4;
	if (index < DM4) {
		// Fill w_i, w_f, w_c and w_o
		int localIndex = index % DM;
		int sdsRowIndex = localIndex / M;
		int sdsColIndex = swap_co(index / (DM)) * M + localIndex % M;
		// Convert index to column-major where index = (index/(DM))*DM + (localIndex/M)*M + localIndex%M
		ret[1] = (index / (DM)) * DM + (localIndex % M) * D + localIndex / M;
		ret[0] = sdsRowIndex * M4 + sdsColIndex;
	} else if (index < (D + M) * M4) {
		// Fill r_i, r_f, r_c and r_o
		int tmpIndex = index - DM4;
		int localIndex = tmpIndex % MM;
		int sdsRowIndex = D + (localIndex / M);
		int sdsColIndex = swap_co(tmpIndex / (MM)) * M + localIndex % M;
		// Convert index to column-major where index = DM4 + (tmpIndex/(MM))*MM + (localIndex/M)*M + localIndex%M
		ret[1] = DM4 + (tmpIndex / (MM)) * MM + (localIndex % M) * M
				+ localIndex / M;
		ret[0] = sdsRowIndex * M4 + sdsColIndex;
	}
}

template<typename T>
__device__ void prepare_lstm_weight(T* sdsWeight, T* sdsBias, T* cudnnWeight,
		int D, int M) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// Maximum (D+M+2)*M4 threads
	int M4 = M * 4;
	if (index < (D + M) * M4) {
		int indexes[2];
		compute_lstm_weight_indexes(index, D, M, indexes);
		cudnnWeight[indexes[1]] = sdsWeight[indexes[0]];
	} else if (index < (D + M + 1) * M4) {
		// Fill bias
		// bias layout: bi bf bc bo 0 0 0 0
		// where W: [DxM], R: [MxM] and b: [1x1]
		int tmpIndex = index - (D + M) * M4;
		int sdsColIndex = swap_co(tmpIndex / (M)) * M + tmpIndex % M;
		cudnnWeight[index] = sdsBias[sdsColIndex];
	}
}

extern "C" __global__ void prepare_lstm_weight_d(double* sdsWeight,
		double* sdsBias, double* cudnnWeight, int D, int M) {
	prepare_lstm_weight(sdsWeight, sdsBias, cudnnWeight, D, M);
}

extern "C" __global__ void prepare_lstm_weight_f(float* sdsWeight,
		float* sdsBias, float* cudnnWeight, int D, int M) {
	prepare_lstm_weight(sdsWeight, sdsBias, cudnnWeight, D, M);
}

// We can later fold it in our reduce method
template<typename T>
__device__ void compute_nnz(T *g_idata, ///< input data stored in device memory (of size n)
		T *g_odata, ///< output/temporary array stored in device memory (of size n)
		unsigned int n)  ///< size of the input and temporary/output arrays
		{
	// extern __shared__ T sdata[];
//  extern __shared__ __align__(sizeof(T)) unsigned char my_sdata[];
//  T *sdata = reinterpret_cast<T *>(my_sdata);
	auto sdata = shared_memory_proxy<T>();

	// perform first level of reduction,
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
	unsigned int gridSize = blockDim.x * 2 * gridDim.x;

	T v = 0;

	// we reduce multiple elements per thread.  The number is determined by the
	// number of active thread blocks (via gridDim).  More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while (i < n) {
		v += g_idata[i] != 0 ? 1 : 0;
		// ensure we don't read out of bounds
		if (i + blockDim.x < n)
			v += g_idata[i + blockDim.x] != 0 ? 1 : 0;
		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = v;
	__syncthreads();

	// do reduction in shared mem
	if (blockDim.x >= 1024) {
		if (tid < 512) {
			sdata[tid] = v = v + sdata[tid + 512];
		}
		__syncthreads();
	}
	if (blockDim.x >= 512) {
		if (tid < 256) {
			sdata[tid] = v = v + sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockDim.x >= 256) {
		if (tid < 128) {
			sdata[tid] = v = v + sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockDim.x >= 128) {
		if (tid < 64) {
			sdata[tid] = v = v + sdata[tid + 64];
		}
		__syncthreads();
	}

	if (tid < 32) {
		// now that we are using warp-synchronous programming (below)
		// we need to declare our shared memory volatile so that the compiler
		// doesn't reorder stores to it and induce incorrect behavior.
		volatile T *smem = sdata;
		if (blockDim.x >= 64) {
			smem[tid] = v = v + smem[tid + 32];
		}
		if (blockDim.x >= 32) {
			smem[tid] = v = v + smem[tid + 16];
		}
		if (blockDim.x >= 16) {
			smem[tid] = v = v + smem[tid + 8];
		}
		if (blockDim.x >= 8) {
			smem[tid] = v = v + smem[tid + 4];
		}
		if (blockDim.x >= 4) {
			smem[tid] = v = v + smem[tid + 2];
		}
		if (blockDim.x >= 2) {
			smem[tid] = v = v + smem[tid + 1];
		}
	}

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

extern "C" __global__ void compute_nnz_d(double *g_idata, double *g_odata,
		unsigned int n) {
	compute_nnz(g_idata, g_odata, n);
}

extern "C" __global__ void compute_nnz_f(float *g_idata, float *g_odata,
		unsigned int n) {
	compute_nnz(g_idata, g_odata, n);
}

template<typename T>
__device__ void prepare_lstm_output(T* sdsInput, T* cudnnInput, int N, int T1,
		int M, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		int TM = T1 * M;
		int n = index / TM;
		int tm = index % TM;
		int t = tm / M;
		int m = tm % M;
		sdsInput[index] = cudnnInput[t * N * M + n * M + m];
	}
}

extern "C" __global__ void prepare_lstm_output_d(double* sdsInput,
		double* cudnnInput, int N, int T, int M, int size) {
	prepare_lstm_output(sdsInput, cudnnInput, N, T, M, size);
}

extern "C" __global__ void prepare_lstm_output_f(float* sdsInput,
		float* cudnnInput, int N, int T, int M, int size) {
	prepare_lstm_output(sdsInput, cudnnInput, N, T, M, size);
}

template<typename T>
__device__ void prepare_lstm_backward_gradients(T* sdsDout, T* cudnnDy, int N,
		int T1, int M, int size, int return_sequences) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size && return_sequences != 0) {
		// sdsDout = [N, T, M]
		int TM = T1 * M;
		int n = index / TM;
		int tm = index % TM;
		int t = tm / M;
		int m = tm % M;
		T val = sdsDout[index];
		cudnnDy[t * N * M + n * M + m] = val;
	} else if (index < size) {
		// sdsDout = [N, T, M]
		int n = index / M;
		int m = index % M;
		T val = sdsDout[index];
		cudnnDy[(T1 - 1) * N * M + n * M + m] = val;
	}
}

extern "C" __global__ void prepare_lstm_backward_gradients_d(double* sdsInput,
		double* cudnnDy, int N, int T, int M, int size, int return_sequences) {
	prepare_lstm_backward_gradients(sdsInput, cudnnDy, N, T, M, size,
			return_sequences);
}

extern "C" __global__ void prepare_lstm_backward_gradients_f(float* sdsInput,
		float* cudnnDy, int N, int T, int M, int size, int return_sequences) {
	prepare_lstm_backward_gradients(sdsInput, cudnnDy, N, T, M, size,
			return_sequences);
}

template<typename T>
__device__ void prepare_lstm_dweight(T* sdsdWeight, T* sdsdBias,
		T* cudnndWeight, int D, int M) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// Maximum (D+M+2)*M4 threads
	int M4 = M * 4;
	if (index < (D + M) * M4) {
		int indexes[2];
		compute_lstm_weight_indexes(index, D, M, indexes);
		sdsdWeight[indexes[0]] = cudnndWeight[indexes[1]];
	} else if (index < (D + M + 1) * M4) {
		// Fill bias
		// bias layout: bi bf bc bo 0 0 0 0
		// where W: [DxM], R: [MxM] and b: [1x1]
		int tmpIndex = index - (D + M) * M4;
		int sdsColIndex = swap_co(tmpIndex / (M)) * M + tmpIndex % M;
		sdsdBias[sdsColIndex] = cudnndWeight[index];
	}
}

extern "C" __global__ void prepare_lstm_dweight_d(double* sdsdWeight,
		double* sdsdBias, double* cudnndWeight, int D, int M) {
	prepare_lstm_dweight(sdsdWeight, sdsdBias, cudnndWeight, D, M);
}

extern "C" __global__ void prepare_lstm_dweight_f(float* sdsdWeight,
		float* sdsdBias, float* cudnndWeight, int D, int M) {
	prepare_lstm_dweight(sdsdWeight, sdsdBias, cudnndWeight, D, M);
}

template<typename T>
__device__ void prepare_lstm_dinput(T* sdsInput, T* cudnnInput, int N, int D,
		int TD, int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		int n = index / TD;
		int td = index % TD;
		int t = td / D;
		int d = td % D;
		sdsInput[index] = cudnnInput[t * N * D + n * D + d];
	}
}

extern "C" __global__ void prepare_lstm_dinput_d(double* sdsInput,
		double* cudnnInput, int N, int D, int TD, int size) {
	prepare_lstm_dinput(sdsInput, cudnnInput, N, D, TD, size);
}

extern "C" __global__ void prepare_lstm_dinput_f(float* sdsInput,
		float* cudnnInput, int N, int D, int TD, int size) {
	prepare_lstm_dinput(sdsInput, cudnnInput, N, D, TD, size);
}

template<typename T>
__device__ void colwise_reshape(T *A, T *C, unsigned int size,
		unsigned int inRows, unsigned int inCols, unsigned int outRows,
		unsigned int outCols) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		int i = index / outCols;
		int j = index % outCols;
		int k = (outRows * j + i) % inRows;
		int l = (outRows * j + i) / inRows;
		C[index] = A[k * inCols + l];
	}
}

extern "C" __global__ void colwise_reshape_d(double *A, double *C,
		unsigned int size, unsigned int inRows, unsigned int inCols,
		unsigned int outRows, unsigned int outCols) {
	colwise_reshape(A, C, size, inRows, inCols, outRows, outCols);
}

extern "C" __global__ void colwise_reshape_f(float *A, float *C,
		unsigned int size, unsigned int inRows, unsigned int inCols,
		unsigned int outRows, unsigned int outCols) {
	colwise_reshape(A, C, size, inRows, inCols, outRows, outCols);
}

// Performs the operation: out = X - mu*v_prev + (1+mu)*v
template<typename T>
__device__ void update_nesterov_x(T *X, T *v, T *v_prev, double mu, T *out,
		unsigned int size) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) {
		out[index] = X[index] - mu * v_prev[index] + (1 + mu) * v[index];
	}
}

extern "C" __global__ void update_nesterov_x_d(double *X, double *v,
		double *v_prev, double mu, double *out, unsigned int size) {
	update_nesterov_x(X, v, v_prev, mu, out, size);
}

extern "C" __global__ void update_nesterov_x_f(float *X, float *v,
		float *v_prev, double mu, float *out, unsigned int size) {
	update_nesterov_x(X, v, v_prev, mu, out, size);
}
